#include "hip/hip_runtime.h"
#include "neural_cache.hpp"

#include "util.h"

#include "../tinyexr_wrapper.h"
#include "../hip/hip_vector_types.h"

#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/config.h>

#include <hip/hip_runtime.h>
#include <cuda_gl_interop.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>
#include <cassert>

#ifndef MAX
#define MAX(a,b) ((a > b) ? a : b)
#endif

using namespace tcnn;
using precision_t = network_precision_t;

static GPUMemory<float> load_image(const std::string &filename, int &width, int &height)
{
	float *out; // width * height * RGBA
	load_exr(&out, &width, &height, filename.c_str());

	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	free(out); // release memory of image data

	return result;
}

//  A key benefit of using the new surface objects is that we don't need any global
//  binding points anymore. We can directly pass them as function arguments.

__global__ void
d_mipmap(hipSurfaceObject_t mipOutput, hipTextureObject_t mipInput, uint32_t imageW, uint32_t imageH)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;

    float px = 1.0/float(imageW);
    float py = 1.0/float(imageH);

    if ((x < imageW) && (y < imageH))
    {
        // take the average of 4 samples

        // we are using the normalized access to make sure non-power-of-two textures
        // behave well when downsized.
        float4 color =
            (tex2D<float4>(mipInput,(x + 0) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput,(x + 1) * px, (y + 0) * py)) +
            (tex2D<float4>(mipInput,(x + 1) * px, (y + 1) * py)) +
            (tex2D<float4>(mipInput,(x + 0) * px, (y + 1) * py));

        color /= 4.0;

        surf2Dwrite(color, mipOutput, x * sizeof(float4), y);
    }
}

void generate_mipmaps(hipMipmappedArray_t mipmapArray, hipExtent size)
{
    size_t width    = size.width;
    size_t height   = size.height;

    uint32_t level = 0;

    while (width != 1 || height != 1)
    {
        width     /= 2;
        width      = MAX((size_t)1, width);
        height    /= 2;
        height     = MAX((size_t)1, height);

        hipArray_t levelFrom;
        CUDA_CHECK_THROW(hipGetMipmappedArrayLevel(&levelFrom, mipmapArray, level));
        hipArray_t levelTo;
        CUDA_CHECK_THROW(hipGetMipmappedArrayLevel(&levelTo,   mipmapArray, level + 1));

        hipExtent levelToSize;
        CUDA_CHECK_THROW(hipArrayGetInfo(NULL, &levelToSize, NULL, levelTo));
        assert(levelToSize.width  == width);
        assert(levelToSize.height == height);
        assert(levelToSize.depth  == 0);

        // generate texture object for reading
        hipResourceDesc texRes;
        memset(&texRes, 0, sizeof(hipResourceDesc));
        texRes.resType         = hipResourceTypeArray;
        texRes.res.array.array = levelFrom;

        hipTextureDesc texDesc;
        memset(&texDesc, 0, sizeof(hipTextureDesc));
        texDesc.filterMode       = hipFilterModeLinear;
        texDesc.normalizedCoords = true;
        texDesc.addressMode[0]   = hipAddressModeClamp;
        texDesc.addressMode[1]   = hipAddressModeClamp;
        texDesc.addressMode[2]   = hipAddressModeClamp;

        hipTextureObject_t texInput;
        CUDA_CHECK_THROW(hipCreateTextureObject(&texInput, &texRes, &texDesc, NULL));

        // generate surface object for writing
        hipResourceDesc surfRes;
        memset(&surfRes,0,sizeof(hipResourceDesc));
        surfRes.resType = hipResourceTypeArray;
        surfRes.res.array.array = levelTo;

        hipSurfaceObject_t surfOutput;
        CUDA_CHECK_THROW(hipCreateSurfaceObject(&surfOutput, &surfRes));

        // run mipmap kernel
        dim3 blockSize(16,16,1);
        dim3 gridSize(((uint32_t)width+blockSize.x-1)/blockSize.x, ((uint32_t)height+blockSize.y-1)/blockSize.y,1);
        d_mipmap<<<gridSize, blockSize>>>(surfOutput, texInput, (uint32_t)width, (uint32_t)height);

        CUDA_CHECK_THROW(hipDeviceSynchronize());
        CUDA_CHECK_THROW(hipGetLastError());
        CUDA_CHECK_THROW(hipDestroySurfaceObject(surfOutput));
        CUDA_CHECK_THROW(hipDestroyTextureObject(texInput));

        level++;
    }
}

uint32_t get_mipmap_levels(hipExtent size)
{
    size_t sz = MAX(MAX(size.width, size.height), size.depth);
    uint32_t levels = 0;
    while (sz)
    {
        sz /= 2;
        levels++;
    }
    return levels;
}

hipTextureObject_t create_mipmap_rgba32f_texture(void* data, int width, int height)
{
    // how many mipmaps we need
    hipExtent extent;
    extent.width  = width;
    extent.height = height;
    extent.depth = 0;
    uint32_t levels = get_mipmap_levels(extent);
        
    hipChannelFormatDesc desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
    hipMipmappedArray_t mipmapArray;
    CUDA_CHECK_THROW(hipMallocMipmappedArray(&mipmapArray, &desc, extent, levels));

    // upload level 0
    hipArray_t level0;
    CUDA_CHECK_THROW(hipGetMipmappedArrayLevel(&level0, mipmapArray, 0));

    hipMemcpy3DParms copyParams = {0};
    copyParams.srcPtr       = make_hipPitchedPtr(data, width * 4 * sizeof(float), width, height);
    copyParams.dstArray     = level0;
    copyParams.extent       = extent;
    copyParams.extent.depth = 1;
    copyParams.kind         = hipMemcpyHostToDevice;
    CUDA_CHECK_THROW(hipMemcpy3D(&copyParams));

    // compute rest of mipmaps based on level 0
    generate_mipmaps(mipmapArray, extent);

    // generate bindless texture object
    hipResourceDesc resDesc;
    memset(&resDesc, 0, sizeof(hipResourceDesc));
    resDesc.resType            = hipResourceTypeMipmappedArray;
    resDesc.res.mipmap.mipmap  = mipmapArray;

    hipTextureDesc texDesc;
    memset(&texDesc, 0, sizeof(hipTextureDesc));
    texDesc.filterMode       = hipFilterModeLinear;
    texDesc.mipmapFilterMode = hipFilterModeLinear;
    texDesc.normalizedCoords = true;
    texDesc.addressMode[0]   = hipAddressModeClamp;
    texDesc.addressMode[1]   = hipAddressModeClamp;
    texDesc.addressMode[2]   = hipAddressModeClamp;
    texDesc.maxMipmapLevelClamp = float(levels - 1);

    hipTextureObject_t texture;
    CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, NULL));
    return texture;
}

hipTextureObject_t create_pitch2d_rgba32f_texture(void* data, int width, int height)
{
    // Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
	hipResourceDesc resDesc;
	memset(&resDesc, 0, sizeof(resDesc));
	resDesc.resType = hipResourceTypePitch2D;
	resDesc.res.pitch2D.devPtr = data;
	resDesc.res.pitch2D.desc = hipCreateChannelDesc(32, 32, 32, 32, hipChannelFormatKindFloat);
	resDesc.res.pitch2D.width = width;
	resDesc.res.pitch2D.height = height;
	resDesc.res.pitch2D.pitchInBytes = width * 4 * sizeof(float);

	hipTextureDesc texDesc;
	memset(&texDesc, 0, sizeof(texDesc));
	texDesc.filterMode = hipFilterModeLinear;
	texDesc.normalizedCoords = true;
	texDesc.addressMode[0] = hipAddressModeClamp;
	texDesc.addressMode[1] = hipAddressModeClamp;
	texDesc.addressMode[2] = hipAddressModeClamp;

	hipResourceViewDesc viewDesc;
	memset(&viewDesc, 0, sizeof(viewDesc));
	viewDesc.format = hipResViewFormatFloat4;
	viewDesc.width = width;
	viewDesc.height = height;

	hipTextureObject_t texture;
	CUDA_CHECK_THROW(hipCreateTextureObject(&texture, &resDesc, &texDesc, &viewDesc));
    return texture;
}

static std::tuple<GPUMemory<float>, hipTextureObject_t> generate_image_texture(std::string filename, int& width, int& height)
{
    // First step: load an image that we'd like to learn
	GPUMemory<float> image = load_image(filename.c_str(), width, height);
    std::cout << "image size: " << width << " " << height << std::endl;

    // Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
	// hipTextureObject_t texture = create_pitch2d_rgba32f_texture(image.data(), width, height);
    hipTextureObject_t texture = create_mipmap_rgba32f_texture(image.data(), width, height);

    return std::make_tuple(std::move(image), texture);
}

template <uint32_t stride>
__global__ void eval_image(uint32_t n_elements, hipTextureObject_t texture, float *__restrict__ xs_and_ys, float *__restrict__ result)
{
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t output_idx = i * stride;
	uint32_t input_idx  = i * 2;

	float4 val = tex2D<float4>(texture, xs_and_ys[input_idx], xs_and_ys[input_idx + 1]);
    
	result[output_idx + 0] = val.x;
	result[output_idx + 1] = val.y;
	result[output_idx + 2] = val.z;

	for (uint32_t i = 3; i < stride; ++i) result[output_idx + i] = 1;
}

__global__ void eval_image_lod(uint32_t n_elements, hipTextureObject_t texture, hipSurfaceObject_t output, int width, int height, float lod)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    float4 color = tex2DLod<float4>(texture, x/(float)width, y/(float)height, lod);
    surf2Dwrite(color, output, x * sizeof(float4), y);
}

struct OpenGLTexture {
private:
    GLuint                 opengl_texture;
    hipGraphicsResource_t cuda_resource_view;

public:
    OpenGLTexture(int width, int height)
    {
        check_error_gl("Create OpenGL Texture");
        glGenTextures(1, &opengl_texture);
        glBindTexture(GL_TEXTURE_2D, opengl_texture); 
        {
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_S, GL_CLAMP_TO_EDGE);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_WRAP_T, GL_CLAMP_TO_EDGE);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MIN_FILTER, GL_LINEAR);
            glTexParameteri(GL_TEXTURE_2D, GL_TEXTURE_MAG_FILTER, GL_LINEAR);
        }
        glBindTexture(GL_TEXTURE_2D, 0);
        check_error_gl("Create OpenGL Texture ... OK");

        resize(width, height);
    }

    ~OpenGLTexture()
    {
        glDeleteTextures(1, &opengl_texture);
    }

    void resize(int width, int height)
    {
        check_error_gl("Resize OpenGL Texture");
        glBindTexture(GL_TEXTURE_2D, opengl_texture); 
        glTexImage2D(GL_TEXTURE_2D, 0, GL_RGBA32F, width, height, 0, GL_RGBA, GL_FLOAT, NULL);
        glBindTexture(GL_TEXTURE_2D, 0);
        check_error_gl("Resize OpenGL Texture ... OK");

        CUDA_CHECK_THROW(hipGraphicsGLRegisterImage(&cuda_resource_view, opengl_texture, GL_TEXTURE_2D, hipGraphicsRegisterFlagsNone));
    }

    void bindOpenGLTexture()
    {
        glBindTexture(GL_TEXTURE_2D, opengl_texture);
        check_error_gl("Bind OpenGL Texture");
    }

    void unbindOpenGLTexture()
    {
        glBindTexture(GL_TEXTURE_2D, 0);
        check_error_gl("Unbind OpenGL Texture");
    }

    void mapCudaArray(hipArray_t& array)
    {
        // We want to copy cuda_dest_resource data to the texture
        // map buffer objects to get CUDA device pointers
        CUDA_CHECK_THROW(hipGraphicsMapResources(1, &cuda_resource_view, 0));
        CUDA_CHECK_THROW(hipGraphicsSubResourceGetMappedArray(&array, cuda_resource_view, 0, 0));
    }

    void unmapCudaArray(hipArray_t&)
    {
        CUDA_CHECK_THROW(hipGraphicsUnmapResources(1, &cuda_resource_view, 0));
    }
};

const uint32_t batch_size = 1 << 12;
const uint32_t n_input_dims  = 2; // 2-D image coordinate
const uint32_t n_output_dims = 4; // RGBA color

struct NeuralImageCache::Impl
{
    typedef std::shared_ptr<OpenGLTexture> openglTextureObject_t;
    typedef GPUMatrix<float, MatrixLayout::ColumnMajor> GPUColumnMatrix;
    
    int width;
    int height;

    GPUMemory<float> xs_and_ys;

    GPUMemory<float> groundtruth_data;
    hipTextureObject_t groundtruth;

    openglTextureObject_t reference_opengl_texture;
    openglTextureObject_t inference_opengl_texture;

    std::unique_ptr<GPUColumnMatrix> inference_input; // Auxiliary matrices for evaluation
    std::unique_ptr<GPUColumnMatrix> inference_result;
    std::unique_ptr<GPUColumnMatrix> training_input;  // Auxiliary matrices for training
    std::unique_ptr<GPUColumnMatrix> training_target;
    hipStream_t inference_stream;
    hipStream_t training_stream;

    std::shared_ptr<Loss<precision_t>> loss;
    std::shared_ptr<Optimizer<precision_t>> optimizer;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;
    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;
    
    hiprandGenerator_t rng;

    float    tmp_loss = 0;
    uint32_t tmp_loss_counter = 0;
    uint64_t total_steps = 0;

    Impl(std::string filename) 
    {
        // Load the image & network configuration
        json config = {
            {
                "loss", {{"otype", "RelativeL2"}}
            }, {
                "optimizer", {
                    {"otype", "Adam"},
                    {"learning_rate", 1e-2},
                    {"beta1", 0.9f},
                    {"beta2", 0.99f},
                    {"epsilon", 1e-8f},
                    {"l2_reg", 1e-8f},
                }
            }, {
                "encoding", {
                    {"otype", "OneBlob"},
                    {"n_bins", 64},
                }
            }, {
                "network", {
                    {"otype", "FullyFusedMLP"},
                    {"n_neurons", 128},
                    {"n_layers", 5},
                    {"activation", "ReLU"},
                    {"output_activation", "None"},
                }
            }
        };
        std::tie(groundtruth_data, groundtruth) = generate_image_texture(filename, width, height);

        uint32_t n_coords = width * height;
        uint32_t n_coords_padded = (n_coords + 255) / 256 * 256;

        // Create the OpenGL texture
        initialize();

        // Generate the coordinates of the image
        std::vector<float> host_xs_and_ys(n_coords * n_input_dims);
        xs_and_ys = GPUMemory<float>(n_coords_padded * n_input_dims);
        for (int y = 0; y < height; ++y)
        {
            for (int x = 0; x < width; ++x)
            {
                int idx = (y * width + x) * n_input_dims;
                host_xs_and_ys[idx + 0] = (float)(x + 0.5) / (float)width;
                host_xs_and_ys[idx + 1] = (float)(y + 0.5) / (float)height;
            }
        }
        xs_and_ys.copy_from_host(host_xs_and_ys.data());

        // Allocate matrices for training and evaluation
        inference_input  = std::make_unique<GPUColumnMatrix>(xs_and_ys.data(), n_input_dims, n_coords_padded);
        inference_result = std::make_unique<GPUColumnMatrix>(n_output_dims, n_coords_padded);
        training_input    = std::make_unique<GPUColumnMatrix>(n_input_dims, batch_size);
        training_target   = std::make_unique<GPUColumnMatrix>(n_output_dims, batch_size);

        // Input & corresponding RNG
        CURAND_CHECK_THROW(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CHECK_THROW(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ULL));

        inference_stream = 0;
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        training_stream = inference_stream;
        CURAND_CHECK_THROW(hiprandSetStream(rng, training_stream));

        // Create the network
        json encoding_opts  = config.value("encoding", json::object());
        json loss_opts      = config.value("loss", json::object());
        json optimizer_opts = config.value("optimizer", json::object());
        json network_opts   = config.value("network", json::object());

        loss = std::shared_ptr<Loss<precision_t>>{create_loss<precision_t>(loss_opts)};
        optimizer = std::shared_ptr<Optimizer<precision_t>>{create_optimizer<precision_t>(optimizer_opts)};
        network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, 0, n_output_dims, encoding_opts, network_opts);
        trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

        // Initialize values
        {
            hipArray_t array;
            reference_opengl_texture->mapCudaArray(array);
            // {
            //     CUDA_CHECK_THROW(hipMemcpyToArray(array, 0, 0, groundtruth_data.data(), sizeof(float) * n_coords * 4, hipMemcpyDeviceToDevice));
            // }
            {
                hipResourceDesc surfRes;
                memset(&surfRes,0,sizeof(hipResourceDesc));
                surfRes.resType = hipResourceTypeArray;
                surfRes.res.array.array = array;

                hipSurfaceObject_t surf;
                CUDA_CHECK_THROW(hipCreateSurfaceObject(&surf, &surfRes));

                dim3 blockSize(16,16,1);
                dim3 gridSize(((uint32_t)width+blockSize.x-1)/blockSize.x, ((uint32_t)height+blockSize.y-1)/blockSize.y,1);
                eval_image_lod<<<gridSize, blockSize>>>(n_coords, groundtruth, surf, width, height, 4.f);
            }
            reference_opengl_texture->unmapCudaArray(array);
        }

        linear_kernel(eval_image<4>, 0, inference_stream, n_coords, groundtruth, inference_input->data(), inference_result->data());
        synchronize(inference_result->data());
    }

    ~Impl()
    {
        reference_opengl_texture.reset();
        inference_opengl_texture.reset();
    }

    void train(size_t steps)
    {
        /* now randomly sample some data */
        for (int i = 0; i < steps; ++i)
        {
            // Third step: sample a reference image to dump to disk. Visual comparison of this reference image and the learned
            //             function will be eventually possible.
            {
                CURAND_CHECK_THROW(hiprandGenerateUniform(rng, training_input->data(), batch_size * n_input_dims));
                linear_kernel(eval_image<n_output_dims>, 0, training_stream, batch_size, groundtruth, training_input->data(), training_target->data());
            }

            float loss_value;
            {
                trainer->training_step(training_stream, *training_input, *training_target, &loss_value);
            }
            tmp_loss += loss_value;
            ++tmp_loss_counter;
        }
        total_steps += steps;
    }
    
    void renderInference()
    {
        network->inference(inference_stream, *inference_input, *inference_result);
        synchronize(inference_result->data());
    }
    
    float currentLoss()
    {
        float ret = tmp_loss / (float)tmp_loss_counter;
        std::cout << "step=" << total_steps << "\tloss=" << ret << std::endl;
        tmp_loss = 0;
        tmp_loss_counter = 0;
        return ret;
    }

    void initialize() 
    {
        reference_opengl_texture = std::make_shared<OpenGLTexture>(width, height);
        inference_opengl_texture = std::make_shared<OpenGLTexture>(width, height);
    }
    
    void synchronize(void* device_ptr)
    {
        // We want to copy cuda_dest_resource data to the texture
        // map buffer objects to get CUDA device pointers
        hipArray_t texture_ptr;
        inference_opengl_texture->mapCudaArray(texture_ptr);
    
        static_assert(n_output_dims == 4);
        int num_of_bytes = sizeof(float) * width * height * 4;
        CUDA_CHECK_THROW(hipMemcpyToArray(texture_ptr, 0, 0, device_ptr, num_of_bytes, hipMemcpyDeviceToDevice));

        inference_opengl_texture->unmapCudaArray(texture_ptr);
    }
};

NeuralImageCache::~NeuralImageCache()
{
    pimpl.reset();
}

NeuralImageCache::NeuralImageCache(std::string filename)
    : pimpl(new Impl(filename))
{
}

void NeuralImageCache::bindInferenceTexture()
{
    pimpl->inference_opengl_texture->bindOpenGLTexture();
}

void NeuralImageCache::bindReferenceTexture()
{
    pimpl->reference_opengl_texture->bindOpenGLTexture();
}

void NeuralImageCache::train(size_t steps)
{
    pimpl->train(steps);
}

void NeuralImageCache::renderInference()
{
    pimpl->renderInference();
}

float NeuralImageCache::currentLoss()
{
    return pimpl->currentLoss();
}
