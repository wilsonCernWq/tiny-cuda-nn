#include "hip/hip_runtime.h"
#include "neuralcache.hpp"

#include "tinyexr_wrapper.h"
#include "helper_cuda_texture.h"
#include "texture_interop.h"
#include "groundtruth_loss.h"

#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/config.h>

#include <hip/hip_runtime.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

#ifndef MIN
#define MIN(a,b) ((a < b) ? a : b)
#endif
#ifndef MAX
#define MAX(a,b) ((a > b) ? a : b)
#endif

using namespace tcnn;
using precision_t = network_precision_t;

static GPUMemory<float> load_image(const std::string &filename, int &width, int &height)
{
	float *out; // width * height * RGBA
	load_exr(&out, &width, &height, filename.c_str());

	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	free(out); // release memory of image data

	return result;
}

static std::tuple<GPUMemory<float>, hipTextureObject_t> generate_image_texture(std::string filename, int& width, int& height)
{
    // First step: load an image that we'd like to learn
	GPUMemory<float> image = load_image(filename.c_str(), width, height);
    // std::cout << "image size: " << width << " " << height << std::endl;

    // Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
    hipTextureObject_t texture = create_mipmap_rgba32f_texture(image.data(), width, height);

    return std::make_tuple(std::move(image), texture);
}

__device__ inline void quantize_sampling_inputs_inner
(
    uint32_t i, uint32_t width, uint32_t height, int lod, float *__restrict__ inputs
)
{
	const uint32_t r = 1 << lod;
	const uint32_t w = width / r;
	const uint32_t h = height / r;
	const uint32_t idx = i * 2;
	const uint32_t x = (1.f - inputs[idx + 0]) * (float)w;
	const uint32_t y = (1.f - inputs[idx + 1]) * (float)h;
	inputs[idx + 0] = (x + 0.5) / (float)w;
	inputs[idx + 1] = (y + 0.5) / (float)h;
}

__global__ void quantize_sampling_inputs_fixed_lod
(
    uint32_t n_elements, uint32_t width, uint32_t height, float *__restrict__ inputs, int lod
)
{
	const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

    if (lod == 0) return;

    quantize_sampling_inputs_inner(i, width, height, lod, inputs);
}

__global__ void quantize_sampling_inputs_variable_lod
(
    uint32_t n_elements, uint32_t width, uint32_t height, float *__restrict__ inputs, int max_lod, float *__restrict__ lods
)
{
 	const uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

    if (max_lod == 0) return;

    const int lod = (1.f - lods[i]) * max_lod;
    quantize_sampling_inputs_inner(i, width, height, lod, inputs);
}

__global__ void sample_tile
(
    uint32_t n_elements, uint32_t n_offset, float4 tile, float *__restrict__ inputs
)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    i += n_offset;

    const uint32_t idx = i * 2;
    const float x = inputs[idx + 0] * tile.z + tile.x;
    const float y = inputs[idx + 1] * tile.w + tile.y;
    inputs[idx + 0] = MIN(x, 1.f);
    inputs[idx + 1] = MIN(y, 1.f);
}

template <uint32_t stride>
__global__ void sample_groundtruth
(
    uint32_t n_elements, uint32_t n_offset, hipTextureObject_t groundtruth, int level_of_detail, 
    float *__restrict__ xs_and_ys, float *__restrict__ results
)
{
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

    i += n_offset;

	const uint32_t output_idx = i * stride;
	const uint32_t input_idx  = i * 2;

    float4 sample = tex2DLod<float4>(groundtruth, xs_and_ys[input_idx], xs_and_ys[input_idx + 1], (float)level_of_detail);

	results[output_idx + 0] = sample.x;
	results[output_idx + 1] = sample.y;
	results[output_idx + 2] = sample.z;
    results[output_idx + 3] = sample.w;

	for (uint32_t i = 4; i < stride; ++i) results[output_idx + i] = 1;
}

template <uint32_t stride>
__global__ void sample_groundtruth_in_tile
(
    uint32_t n_elements, uint32_t n_offset, hipTextureObject_t groundtruth, int level_of_detail, float4 tile,
    float *__restrict__ xs_and_ys, float *__restrict__ results
)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= n_elements) return;

    i += n_offset;

    const uint32_t input_idx = i * 2;
    const float x = xs_and_ys[input_idx];
    const float y = xs_and_ys[input_idx + 1];
    if (x <  tile.x) return;
    if (y <  tile.y) return;
    if (x >= tile.x + tile.z) return;
    if (y >= tile.y + tile.w) return;
    float4 sample = tex2DLod<float4>(groundtruth, x, y, (float)level_of_detail);

    const uint32_t output_idx = i * stride;
    results[output_idx + 0] = sample.x;
    results[output_idx + 1] = sample.y;
    results[output_idx + 2] = sample.z;
    results[output_idx + 3] = sample.w;

    for (uint32_t i = 4; i < stride; ++i) results[output_idx + i] = 1;
}

__global__ void resample_texture_with_lod
(
    uint32_t width, uint32_t height, hipTextureObject_t texture, hipSurfaceObject_t output, int lod
)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    float4 color = tex2DLod<float4>(texture, x/(float)width, y/(float)height, (float)lod);
    surf2Dwrite(color, output, x * sizeof(float4), y);
}

#define LOSS_INNER relative_l2_loss

template <uint32_t stride>
__global__ void compute_groundtruth_loss
(    
    uint32_t n_elements, uint32_t n_offset, hipTextureObject_t groundtruth,
    float *__restrict__ xs_and_ys, float *__restrict__ predictions, float *__restrict__ values
)
{
    uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

    i += n_offset;

	const uint32_t input_idx  = i * 2;
    const uint32_t output_idx = i * stride;

    float4 target = tex2D<float4>(groundtruth, xs_and_ys[input_idx], xs_and_ys[input_idx + 1]);

	float v = LOSS_INNER(predictions[output_idx + 0], target.x);
    v += LOSS_INNER(predictions[output_idx + 1], target.y);
    v += LOSS_INNER(predictions[output_idx + 2], target.z);
    v += LOSS_INNER(predictions[output_idx + 3], target.w);
    for (uint32_t i = 4; i < stride; ++i) v += LOSS_INNER(predictions[output_idx + i], 1.f);

	values[i] = v / n_elements / 4;
}

const uint32_t batch_size = 1 << 20;
const uint32_t n_input_dims = 2; // 2-D image coordinate
const uint32_t n_output_dims = 4; // RGBA color

struct NeuralImageCache::Impl
{
    typedef std::shared_ptr<OpenGLTexture> openglTextureObject_t;
    typedef GPUMatrix<float, MatrixLayout::ColumnMajor> GPUColumnMatrix;

    int width;
    int height;

    GPUMemory<float> xs_and_ys;

    GPUMemory<float> groundtruth_data;
    hipTextureObject_t groundtruth;

    openglTextureObject_t reference_opengl_texture;
    openglTextureObject_t inference_opengl_texture;

    std::unique_ptr<GPUColumnMatrix> inference_input; // Auxiliary matrices for evaluation
    std::unique_ptr<GPUColumnMatrix> inference_result;
    std::unique_ptr<GPUColumnMatrix> training_input;  // Auxiliary matrices for training
    std::unique_ptr<GPUColumnMatrix> training_target;

    hipStream_t inference_stream;
    hipStream_t training_stream;

    std::shared_ptr<Loss<precision_t>> loss;
    std::shared_ptr<Optimizer<precision_t>> optimizer;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;
    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;

    default_rng_t rng{1337};

    std::unique_ptr<GPUColumnMatrix> groundtruth_loss_values;
    float training_loss = 0;
    uint32_t training_loss_counter = 0;

    uint64_t training_step = 0;

    int level_of_detail = 0;
    int tile_size = 128;
    int2 tile_dims;
    float2 tile_scale;

    Impl(std::string filename) 
    {
        // Load the image & network configuration
        json config = {
            {
                "loss", {{"otype", "RelativeL2"}}
            }, {
                "optimizer", {
                    {"otype", "Adam"},
                    {"learning_rate", 1e-2},
                    {"beta1", 0.9f},
                    {"beta2", 0.999f},
                    {"epsilon", 1e-8f},
                    {"l2_reg", 1e-8f},
                }
            }, {
                "encoding", {
                    {"otype", "OneBlob"},
                    {"n_bins", 64},
                }
            }, {
                "network", {
                    {"otype", "FullyFusedMLP"},
                    {"n_neurons", 128},
                    {"n_layers", 5},
                    {"activation", "ReLU"},
                    {"output_activation", "None"},
                }
            }
        };
        std::tie(groundtruth_data, groundtruth) = generate_image_texture(filename, width, height);
        updateTileDimensions();

        uint32_t n_coords = width * height;
        uint32_t n_coords_padded = (n_coords + 255) / 256 * 256;

        // Create the OpenGL texture
        initialize();

        // Generate the coordinates of the image
        std::vector<float> host_xs_and_ys(n_coords * n_input_dims);
        xs_and_ys = GPUMemory<float>(n_coords_padded * n_input_dims);
        for (int y = 0; y < height; ++y)
        {
            for (int x = 0; x < width; ++x)
            {
                int idx = (y * width + x) * n_input_dims;
                host_xs_and_ys[idx + 0] = (float)(x + 0.5) / (float)width;
                host_xs_and_ys[idx + 1] = (float)(y + 0.5) / (float)height;
            }
        }
        xs_and_ys.copy_from_host(host_xs_and_ys.data());

        // Allocate matrices for training and evaluation
        inference_input = std::make_unique<GPUColumnMatrix>(xs_and_ys.data(), n_input_dims, n_coords_padded);
        inference_result = std::make_unique<GPUColumnMatrix>(n_output_dims, n_coords_padded);
        training_input = std::make_unique<GPUColumnMatrix>(n_input_dims, batch_size);
        training_target = std::make_unique<GPUColumnMatrix>(n_output_dims, batch_size);

        groundtruth_loss_values = std::make_unique<GPUColumnMatrix>(1, n_coords_padded);

        // Input
        inference_stream = 0;
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        training_stream = inference_stream;

        // Create the network
        json encoding_opts = config.value("encoding", json::object());
        json loss_opts = config.value("loss", json::object());
        json optimizer_opts = config.value("optimizer", json::object());
        json network_opts = config.value("network", json::object());

        loss = std::shared_ptr<Loss<precision_t>>{create_loss<precision_t>(loss_opts)};
        optimizer = std::shared_ptr<Optimizer<precision_t>>{create_optimizer<precision_t>(optimizer_opts)};
        network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, n_output_dims, encoding_opts, network_opts);
        trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

        // Initialize values
        renderInference();
        renderReference();
    }

    ~Impl()
    {
        reference_opengl_texture.reset();
        inference_opengl_texture.reset();
    }

    void updateTileDimensions()
    {
        tile_dims.x = (width  + tile_size - 1) / tile_size;
        tile_dims.y = (height + tile_size - 1) / tile_size;
        tile_scale.x = tile_size / (float)width;
        tile_scale.y = tile_size / (float)height;
    }

    void train(size_t steps, SamplingMode mode)
    {
        /* now randomly sample some data */
        for (int i = 0; i < steps; ++i)
        {
            // Third step: sample a reference image to dump to disk. Visual comparison of this reference image and the learned
            //             function will be eventually possible.

            generate_random_uniform<float>(training_stream, rng, batch_size * n_input_dims, training_input->data());

            const uint32_t tile_index = training_step % ((size_t)tile_dims.x * tile_dims.y);
            float4 tile = make_float4( 
                (tile_index % tile_dims.x) * tile_scale.x,
                (tile_index / tile_dims.y) * tile_scale.y,
                tile_scale.x,
                tile_scale.y
            );

            switch (mode) {
                case UNIFORM_RANDOM: {
                    linear_kernel(sample_groundtruth<n_output_dims>, 0, training_stream, batch_size, 
                        0, groundtruth, level_of_detail, training_input->data(), training_target->data());
                    break;
                }

                case UNIFORM_RANDOM_QUANTIZED: {
                    linear_kernel(quantize_sampling_inputs_fixed_lod, 0, training_stream, batch_size, 
                        width, height, training_input->data(), level_of_detail);
                    linear_kernel(sample_groundtruth<n_output_dims>, 0, training_stream, batch_size, 
                        0, groundtruth, /*level_of_detail=*/0, training_input->data(), training_target->data());
                    break;
                }

                case TILE_BASED_SIMPLE: {
                    linear_kernel(sample_tile, 0, training_stream, batch_size, /*offset=*/0, tile, training_input->data());
                    linear_kernel(sample_groundtruth<n_output_dims>, 0, training_stream, batch_size, 
                        0, groundtruth, level_of_detail, training_input->data(), training_target->data());
                    break;
                }

                case TILE_BASED_MIXTURE: {
                    network->inference(inference_stream, *training_input, *training_target);

                    const float ratio = 0.5f;
                    const auto n_samples = batch_size * ratio;

                    linear_kernel(sample_tile, 0, training_stream, n_samples, 
                        /*offset=*/batch_size - n_samples, tile, training_input->data());

                    linear_kernel(sample_groundtruth<n_output_dims>, 0, training_stream, n_samples, 
                        /*offset=*/batch_size - n_samples, groundtruth, level_of_detail, training_input->data(), training_target->data());
                    break;
                }

                case TILE_BASED_EVENLY: {
                    network->inference(inference_stream, *training_input, *training_target);
                    linear_kernel(sample_groundtruth_in_tile<n_output_dims>, 0, training_stream, batch_size, 
                        /*offset=*/0, groundtruth, level_of_detail, tile, training_input->data(), training_target->data());
                    break;
                }
            }

            float loss_value;
            {
                trainer->training_step(training_stream, *training_input, *training_target, &loss_value);
            }
            training_loss += loss_value;
            ++training_loss_counter;

            ++training_step;
        }
    }

    void renderInference()
    {
        network->inference(inference_stream, *inference_input, *inference_result);

        // We want to copy cuda_dest_resource data to the texture
        // map buffer objects to get CUDA device pointers
        hipArray_t array;
        inference_opengl_texture->mapCudaArray(array);
        {
            CUDA_CHECK_THROW(hipMemcpyToArray(array, 0, 0, inference_result->data(), sizeof(float) * width * height * 4, hipMemcpyDeviceToDevice));
        }
        inference_opengl_texture->unmapCudaArray(array);
    }

    void renderReference()
    {
        // We want to copy cuda_dest_resource data to the texture
        // map buffer objects to get CUDA device pointers
        hipArray_t array;
        reference_opengl_texture->mapCudaArray(array);
        /* show ground truth at the full resolution */
        // CUDA_CHECK_THROW(hipMemcpyToArray(array, 0, 0, groundtruth_data.data(), sizeof(float) * width * height * 4, hipMemcpyDeviceToDevice));
        /* show ground truth with lod */
        {
            hipResourceDesc resDesc;
            memset(&resDesc,0,sizeof(hipResourceDesc));
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = array;
            hipSurfaceObject_t arraySurf;
            CUDA_CHECK_THROW(hipCreateSurfaceObject(&arraySurf, &resDesc));
            bilinear_kernel(resample_texture_with_lod, 0, /*stream=*/inference_stream, width, height, groundtruth, arraySurf, /*lod=*/level_of_detail);
        }
        reference_opengl_texture->unmapCudaArray(array);
    }

    float trainingLoss()
    {   
        float ret = training_loss / (float)training_loss_counter;
        training_loss = 0;
        training_loss_counter = 0;
        return ret;
    }

    float groundtruthLoss()
    {
        linear_kernel(compute_groundtruth_loss<n_output_dims>, 0, inference_stream, groundtruth_loss_values->n_elements(), 
            /*offset=*/0, groundtruth, inference_input->data(), inference_result->data(), groundtruth_loss_values->data());
        return reduce_sum(groundtruth_loss_values->data(), groundtruth_loss_values->n_elements(), inference_stream);
    }

    void initialize() 
    {
        reference_opengl_texture = std::make_shared<OpenGLTexture>(width, height);
        inference_opengl_texture = std::make_shared<OpenGLTexture>(width, height);
    }

};

NeuralImageCache::~NeuralImageCache()
{
    pimpl.reset();
}

NeuralImageCache::NeuralImageCache(std::string filename)
    : pimpl(new Impl(filename))
{
}

void NeuralImageCache::bindInferenceTexture()
{
    pimpl->inference_opengl_texture->bindOpenGLTexture();
}

void NeuralImageCache::bindReferenceTexture()
{
    pimpl->reference_opengl_texture->bindOpenGLTexture();
}

void NeuralImageCache::setLod(int level_of_detail)
{
    pimpl->level_of_detail = level_of_detail;
}

void NeuralImageCache::setTileSize(int tile_size)
{
    pimpl->tile_size = tile_size;
    pimpl->updateTileDimensions();
}

void NeuralImageCache::train(size_t steps, SamplingMode mode)
{
    pimpl->train(steps, mode);
}

void NeuralImageCache::renderInference()
{
    pimpl->renderInference();
}

void NeuralImageCache::renderReference()
{
    pimpl->renderReference();
}

void NeuralImageCache::trainingStats(size_t steps, float& training_loss, float& groundtruth_loss)
{
    steps = pimpl->training_step;
    training_loss = pimpl->trainingLoss();
    groundtruth_loss = pimpl->groundtruthLoss();
    std::cout << "step=" << steps << "\ttloss=" << training_loss << "\tgloss=" << groundtruth_loss <<std::endl;
}
