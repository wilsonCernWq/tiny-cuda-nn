#include "hip/hip_runtime.h"
#include "neuralcache.hpp"

#include "tinyexr_wrapper.h"
#include "helper_cuda_texture.h"
#include "texture_interop.h"

#include <tiny-cuda-nn/misc_kernels.h>
#include <tiny-cuda-nn/config.h>

#include <hip/hip_runtime.h>

#include <chrono>
#include <cstdlib>
#include <fstream>
#include <iostream>
#include <random>
#include <stdexcept>
#include <string>
#include <thread>
#include <vector>

#ifndef MIN
#define MIN(a,b) ((a < b) ? a : b)
#endif

using namespace tcnn;
using precision_t = network_precision_t;

//  Work with OpenEXR images

static GPUMemory<float> load_image(const std::string &filename, int &width, int &height)
{
	float *out; // width * height * RGBA
	load_exr(&out, &width, &height, filename.c_str());

	GPUMemory<float> result(width * height * 4);
	result.copy_from_host(out);
	free(out); // release memory of image data

	return result;
}

static std::tuple<GPUMemory<float>, hipTextureObject_t> generate_image_texture(std::string filename, int& width, int& height)
{
    // First step: load an image that we'd like to learn
	GPUMemory<float> image = load_image(filename.c_str(), width, height);
    std::cout << "image size: " << width << " " << height << std::endl;

    // Second step: create a cuda texture out of this image. It'll be used to generate training data efficiently on the fly
	// hipTextureObject_t texture = create_pitch2d_rgba32f_texture(image.data(), width, height);
    hipTextureObject_t texture = create_mipmap_rgba32f_texture(image.data(), width, height);

    return std::make_tuple(std::move(image), texture);
}

__device__ inline void quantize_sampling_inputs_inner(uint32_t i, uint32_t width, uint32_t height, int lod, float *__restrict__ inputs)
{
	const uint32_t r = 1 << lod;
	const uint32_t w = width / r;
	const uint32_t h = height / r;

	const uint32_t idx = i * 2;

	const uint32_t x = (1.f - inputs[idx + 0]) * (float)w;
	const uint32_t y = (1.f - inputs[idx + 1]) * (float)h;
	inputs[idx + 0] = (x + 0.5) / (float)w;
	inputs[idx + 1] = (y + 0.5) / (float)h;
}

__global__ void quantize_sampling_inputs_fixed_lod(uint32_t n_elements, uint32_t width, uint32_t height, int lod, float *__restrict__ inputs)
{
    if (lod == 0) return;

	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

    quantize_sampling_inputs_inner(i, width, height, lod, inputs);
}

__global__ void quantize_sampling_inputs_variable_lod(uint32_t n_elements, uint32_t width, uint32_t height, int max_lod, float *__restrict__ lods, float *__restrict__ inputs)
{
    if (max_lod == 0) return;

	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	int lod = (1.f - lods[i]) * max_lod;
    quantize_sampling_inputs_inner(i, width, height, lod, inputs);
}

template <uint32_t stride>
__global__ void sample_groundtruth(uint32_t n_elements, hipTextureObject_t groundtruth, float *__restrict__ xs_and_ys, float *__restrict__ result)
{
	uint32_t i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i >= n_elements) return;

	uint32_t output_idx = i * stride;
	uint32_t input_idx  = i * 2;

	float4 sample = tex2D<float4>(groundtruth, xs_and_ys[input_idx], xs_and_ys[input_idx + 1]);

	result[output_idx + 0] = sample.x;
	result[output_idx + 1] = sample.y;
	result[output_idx + 2] = sample.z;
    result[output_idx + 3] = sample.w;

	for (uint32_t i = 4; i < stride; ++i) result[output_idx + i] = 1;
}

__global__ void resample_texture_with_lod(uint32_t width, uint32_t height, hipTextureObject_t texture, hipSurfaceObject_t output, int lod)
{
    uint32_t x = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t y = blockIdx.y * blockDim.y + threadIdx.y;
    float4 color = tex2DLod<float4>(texture, x/(float)width, y/(float)height, (float)lod);
    surf2Dwrite(color, output, x * sizeof(float4), y);
}

const uint32_t batch_size = 1 << 12;
const uint32_t n_input_dims  = 2; // 2-D image coordinate
const uint32_t n_output_dims = 4; // RGBA color

struct NeuralImageCache::Impl
{
    typedef std::shared_ptr<OpenGLTexture> openglTextureObject_t;
    typedef GPUMatrix<float, MatrixLayout::ColumnMajor> GPUColumnMatrix;
    
    int width;
    int height;

    GPUMemory<float> xs_and_ys;

    GPUMemory<float> groundtruth_data;
    hipTextureObject_t groundtruth;

    openglTextureObject_t reference_opengl_texture;
    openglTextureObject_t inference_opengl_texture;

    std::unique_ptr<GPUColumnMatrix> inference_input; // Auxiliary matrices for evaluation
    std::unique_ptr<GPUColumnMatrix> inference_result;
    std::unique_ptr<GPUColumnMatrix> training_input;  // Auxiliary matrices for training
    std::unique_ptr<GPUColumnMatrix> training_target;
    hipStream_t inference_stream;
    hipStream_t training_stream;

    std::shared_ptr<Loss<precision_t>> loss;
    std::shared_ptr<Optimizer<precision_t>> optimizer;
    std::shared_ptr<NetworkWithInputEncoding<precision_t>> network;
    std::shared_ptr<Trainer<float, precision_t, precision_t>> trainer;
    
    hiprandGenerator_t rng;

    float    tmp_loss = 0;
    uint32_t tmp_loss_counter = 0;
    uint64_t total_steps = 0;

    int level_of_detail = 0;

    Impl(std::string filename) 
    {
        // Load the image & network configuration
        json config = {
            {
                "loss", {{"otype", "RelativeL2"}}
            }, {
                "optimizer", {
                    {"otype", "Adam"},
                    {"learning_rate", 1e-2},
                    {"beta1", 0.9f},
                    {"beta2", 0.99f},
                    {"epsilon", 1e-8f},
                    {"l2_reg", 1e-8f},
                }
            }, {
                "encoding", {
                    {"otype", "OneBlob"},
                    {"n_bins", 64},
                }
            }, {
                "network", {
                    {"otype", "FullyFusedMLP"},
                    {"n_neurons", 128},
                    {"n_layers", 5},
                    {"activation", "ReLU"},
                    {"output_activation", "None"},
                }
            }
        };
        std::tie(groundtruth_data, groundtruth) = generate_image_texture(filename, width, height);

        uint32_t n_coords = width * height;
        uint32_t n_coords_padded = (n_coords + 255) / 256 * 256;

        // Create the OpenGL texture
        initialize();

        // Generate the coordinates of the image
        std::vector<float> host_xs_and_ys(n_coords * n_input_dims);
        xs_and_ys = GPUMemory<float>(n_coords_padded * n_input_dims);
        for (int y = 0; y < height; ++y)
        {
            for (int x = 0; x < width; ++x)
            {
                int idx = (y * width + x) * n_input_dims;
                host_xs_and_ys[idx + 0] = (float)(x + 0.5) / (float)width;
                host_xs_and_ys[idx + 1] = (float)(y + 0.5) / (float)height;
            }
        }
        xs_and_ys.copy_from_host(host_xs_and_ys.data());

        // Allocate matrices for training and evaluation
        inference_input  = std::make_unique<GPUColumnMatrix>(xs_and_ys.data(), n_input_dims, n_coords_padded);
        inference_result = std::make_unique<GPUColumnMatrix>(n_output_dims, n_coords_padded);
        training_input   = std::make_unique<GPUColumnMatrix>(n_input_dims, batch_size);
        training_target  = std::make_unique<GPUColumnMatrix>(n_output_dims, batch_size);

        // Input & corresponding RNG
        CURAND_CHECK_THROW(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
        CURAND_CHECK_THROW(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ULL));

        inference_stream = 0;
        CUDA_CHECK_THROW(hipStreamCreate(&inference_stream));
        training_stream = inference_stream;
        CURAND_CHECK_THROW(hiprandSetStream(rng, training_stream));

        // Create the network
        json encoding_opts  = config.value("encoding", json::object());
        json loss_opts      = config.value("loss", json::object());
        json optimizer_opts = config.value("optimizer", json::object());
        json network_opts   = config.value("network", json::object());

        loss = std::shared_ptr<Loss<precision_t>>{create_loss<precision_t>(loss_opts)};
        optimizer = std::shared_ptr<Optimizer<precision_t>>{create_optimizer<precision_t>(optimizer_opts)};
        network = std::make_shared<NetworkWithInputEncoding<precision_t>>(n_input_dims, 0, n_output_dims, encoding_opts, network_opts);
        trainer = std::make_shared<Trainer<float, precision_t, precision_t>>(network, optimizer, loss);

        // Initialize values
        renderInference();
        renderReference();
    }

    ~Impl()
    {
        reference_opengl_texture.reset();
        inference_opengl_texture.reset();
    }

    void train(size_t steps)
    {
        /* now randomly sample some data */
        for (int i = 0; i < steps; ++i)
        {
            // Third step: sample a reference image to dump to disk. Visual comparison of this reference image and the learned
            //             function will be eventually possible.
            {
                CURAND_CHECK_THROW(hiprandGenerateUniform(rng, training_input->data(), batch_size * n_input_dims));
                linear_kernel(quantize_sampling_inputs_fixed_lod, 0, training_stream, batch_size, width, height, level_of_detail, training_input->data());
                linear_kernel(sample_groundtruth<n_output_dims>, 0, training_stream, batch_size, groundtruth, training_input->data(), training_target->data());
            }

            float loss_value;
            {
                trainer->training_step(training_stream, *training_input, *training_target, &loss_value);
            }
            tmp_loss += loss_value;
            ++tmp_loss_counter;
        }
        total_steps += steps;
    }
    
    void renderInference()
    {
        network->inference(inference_stream, *inference_input, *inference_result);

        // linear_kernel(sample_groundtruth<n_output_dims>, 0, inference_stream, width * height, groundtruth, inference_input->data(), inference_result->data());

        // We want to copy cuda_dest_resource data to the texture
        // map buffer objects to get CUDA device pointers
        hipArray_t array;
        inference_opengl_texture->mapCudaArray(array);
        {
            CUDA_CHECK_THROW(hipMemcpyToArray(array, 0, 0, inference_result->data(), sizeof(float) * width * height * 4, hipMemcpyDeviceToDevice));
        }
        inference_opengl_texture->unmapCudaArray(array);
    }

    void renderReference()
    {
        // We want to copy cuda_dest_resource data to the texture
        // map buffer objects to get CUDA device pointers
        hipArray_t array;
        reference_opengl_texture->mapCudaArray(array);
        /* show ground truth at the full resolution */
        // {
        //     CUDA_CHECK_THROW(hipMemcpyToArray(array, 0, 0, groundtruth_data.data(), sizeof(float) * width * height * 4, hipMemcpyDeviceToDevice));
        // }
        /* show ground truth with lod */
        {
            hipResourceDesc resDesc;
            memset(&resDesc,0,sizeof(hipResourceDesc));
            resDesc.resType = hipResourceTypeArray;
            resDesc.res.array.array = array;
            hipSurfaceObject_t arraySurf;
            CUDA_CHECK_THROW(hipCreateSurfaceObject(&arraySurf, &resDesc));
            bilinear_kernel(resample_texture_with_lod, 0, /*stream=*/inference_stream, width, height, groundtruth, arraySurf, /*lod=*/level_of_detail);
        }
        reference_opengl_texture->unmapCudaArray(array);
    }

    float currentLoss()
    {
        float ret = tmp_loss / (float)tmp_loss_counter;
        // std::cout << "step=" << total_steps << "\tloss=" << ret << std::endl;
        tmp_loss = 0;
        tmp_loss_counter = 0;
        return ret;
    }

    void initialize() 
    {
        reference_opengl_texture = std::make_shared<OpenGLTexture>(width, height);
        inference_opengl_texture = std::make_shared<OpenGLTexture>(width, height);
    }

};

NeuralImageCache::~NeuralImageCache()
{
    pimpl.reset();
}

NeuralImageCache::NeuralImageCache(std::string filename)
    : pimpl(new Impl(filename))
{
}

void NeuralImageCache::bindInferenceTexture()
{
    pimpl->inference_opengl_texture->bindOpenGLTexture();
}

void NeuralImageCache::bindReferenceTexture()
{
    pimpl->reference_opengl_texture->bindOpenGLTexture();
}

void NeuralImageCache::setLod(int lod)
{
    pimpl->level_of_detail = lod;
}

void NeuralImageCache::train(size_t steps)
{
    pimpl->train(steps);
}

void NeuralImageCache::renderInference()
{
    pimpl->renderInference();
}

void NeuralImageCache::renderReference()
{
    pimpl->renderReference();
}

float NeuralImageCache::currentLoss()
{
    return pimpl->currentLoss();
}
